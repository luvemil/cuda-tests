#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <gudhi/reader_utils.h>
#include <vector>
#include <algorithm>

#define PT_SIZE 8

// define the types
typedef int Vertex_handle;
typedef double Filtration_value;
typedef std::pair< Vertex_handle, Vertex_handle > Edge_t;
typedef std::vector<double> Point_t;

typedef std::vector< Edge_t > Edges_v;
typedef std::vector< Filtration_value > Filtration_values_v;
typedef std::vector< Point_t > PointCloud;

__global__ void parallel_distance(float **Points, int pt_dim, int pt_n, int *edges,
  double *filt_value, float threshold_q)
{
  int idx = threadIdx.x;
  float me[PT_SIZE];
  for(int i=0; i<pt_dim; i++)
    me[i] = Points[idx*pt_dim + i];

  for(int t=idx+1; t<pt_n; t++)
  {
    float dist_q = 0;
    for(int i=0; i<pt_dim; i++)
    {
      dist_q += fabs(me[i] - Points[t][i]);
      if(dist_q > threshold_q)
      {
        dist_q = -1;
        break;
      }
    }

    if(dist_q > 0)
    {
      int ix = idx * pt_dim + t - 1;
      edges[ix] = 1;
      filt_value[ix] = sqrt(dist_q);
    }
  }
}

void compute_distance_matrix(Edges_v &edges, Filtration_values_v &edges_fil,
  PointCloud &points, Filtration_value threshold)
{
}

int main()
{
  PointCloud points;
  read_points("/home/marco/repos/gudhi/2016-04-15-17-35-51_GUDHI_1.3.0/data/points/Kl.txt",points);

  return 0;
}
